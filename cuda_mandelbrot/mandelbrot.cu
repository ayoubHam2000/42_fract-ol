#include "hip/hip_runtime.h"
# include <stdio.h>
# include <unistd.h>
# include <stdlib.h>
# include <math.h>
# include <SDL2/SDL.h>
# include <hip/hip_runtime.h>
# include <sys/time.h>

# define ZOOM 0.9
# define TRAN 0.2
# define TITLE "Fract"

void	set_colors(int *map)
{
	map[0] = 0x0009012f;
	map[1] = 0x00040449;
	map[2] = 0x00000764;
	map[3] = 0x000c2c8a;
	map[4] = 0x001852b1;
	map[5] = 0x00397dd1;
	map[6] = 0x0086b5e5;
	map[7] = 0x00d3ecf8;
	map[8] = 0x00f1e9bf;
	map[9] = 0x00f8c95f;
	map[10] = 0x00ffaa00;
	map[11] = 0x00cc8000;
	map[12] = 0x00995700;
	map[13] = 0x006a3403;
	map[14] = 0x00421e0f;
	map[15] = 0x0019071a;
}

typedef double t_double;

struct t_complex
{
	double	a;
	double	b;
};

struct GPU_Surface
{
	int 	w;
	int 	h;
	int 	pitch;
	void	*pixels;
	Uint8 BytesPerPixel;
	size_t pixel_size;
};


struct t_global
{
	int						w_size;
  int         	depth;
  t_complex 		x_bound;
	t_complex 		y_bound;
	t_double  		step_x;
	t_double  		step_y;

	int						color_map[16];
	int 					*d_color_map;
  GPU_Surface 	gpu_surface;
	SDL_Window		*win;
	SDL_Surface   *win_surface;
	void					*f;
	t_double  		zoom;
	int						m_wheel;
	int						m_x;
	int						m_y;
	int						quit;
};

struct GPUData {
	int						w_size;
  int         	depth;
  t_complex 		x_bound;
	t_complex 		y_bound;
	t_double  		step_x;
	t_double  		step_y;
};

void cpu_ft_put_pixel(SDL_Surface* surface, unsigned int x, unsigned int y, unsigned int color)
{
	if (y >= (unsigned int)surface->h || x >= (unsigned int)surface->w)
			return ;
	*((Uint32*)((Uint8*)surface->pixels + y * surface->pitch + x * surface->format->BytesPerPixel)) = color;
}

__device__ void ft_put_pixel(GPU_Surface surface, unsigned int x, unsigned int y, unsigned int color)
{
	if (y >= (unsigned int)surface.h || x >= (unsigned int)surface.w)
			return ;
	*((Uint32*)((Uint8*)surface.pixels + y * surface.pitch + x * surface.BytesPerPixel)) = color;
}


__device__ void	trans_iter(double a, double b, int *iter)
{
	t_double	log_zn;
	t_double	nu;

	log_zn = log(a * a + b * b) / 2;
	nu = log(log_zn / __logf(2)) / __logf(2);
	*iter = *iter + 1 - nu;
}

__device__ int	mandelbrot_set(int depth, t_double X, t_double Y)
{
	int				iter;
	double		xtemp;
  double 		a = 0.0;
  double 		b = 0.0;

  iter = 0;
  while (a * a + b * b <= 4 && iter < depth)
  {
    xtemp = a * a - b * b + X;
    b = 2 * a * b + Y;
    a = xtemp;
    iter++;
  }
  if (iter < depth)
	{
    trans_iter(a, b, &iter);
		return (iter % 16);
	}
	return 0;
}


__global__ void	graph_loop(unsigned int *ptr, GPUData data, int start_x, int start_y, int *map)
{
	int x = blockDim.x * blockIdx.x	+ threadIdx.x + start_x;
	int y = blockDim.y * blockIdx.y	+ threadIdx.y + start_y;
	


	if (x >= data.w_size || y >= data.w_size)
		return;

	t_double X = data.step_x * x + data.x_bound.a;
	t_double Y = data.step_y * y + data.y_bound.a;
	int color = mandelbrot_set(data.depth, X, Y);
	ptr[x + y * data.w_size] = map[color];
}


// unsigned int *d_pixels;
// int size = 4 * 1000 * 1000;
// hipMalloc(&d_pixels, size);

void	fractol_frame(t_global *global)
{
	unsigned int *d_pixels = (unsigned int*)(global->gpu_surface.pixels);
	
	// hipStream_t streams[4];
	// hipStreamCreate(&streams[0]);
	// hipStreamCreate(&streams[1]);
	// hipStreamCreate(&streams[2]);
	// hipStreamCreate(&streams[3]);

	GPUData gpu_data = *((GPUData *)global);
	// graph_loop<<<dim3(16, 16), dim3(32, 32), 0, streams[0]>>>(d_pixels, gpu_data, 0, 0, global->d_color_map);
	// graph_loop<<<dim3(16, 16), dim3(32, 32), 0, streams[1]>>>(d_pixels, gpu_data, 512, 0, global->d_color_map);
	// graph_loop<<<dim3(16, 16), dim3(32, 32), 0, streams[2]>>>(d_pixels, gpu_data, 0, 512, global->d_color_map);
	// graph_loop<<<dim3(16, 16), dim3(32, 32), 0, streams[3]>>>(d_pixels, gpu_data, 512, 512, global->d_color_map);
	graph_loop<<<dim3(32, 32), dim3(32, 32)>>>(d_pixels, gpu_data, 0, 0, global->d_color_map);
	hipDeviceSynchronize();


	//update
	hipError_t err = hipSuccess;
	err = hipMemcpy(global->win_surface->pixels, d_pixels, 4 * global->w_size * global->w_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
    printf("Failed (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
	}
	
	SDL_UpdateWindowSurface(global->win);
}

inline void on_mouse_wheel(int y, t_global *global)
{
	global->m_wheel = y;
}

inline void on_mouse_move(int x, int y, t_global *global)
{
	global->m_x = x;
	global->m_y = y;
}

void perform_events(t_global *global)
{
	if (global->m_wheel)
	{
		global->zoom += ZOOM * global->m_wheel * global->zoom;

		// double dir_x = TRAN * (1.0 * global->m_x / global->w_size - 0.5); 
		// double dir_y = TRAN * (1.0 * global->m_y / global->w_size - 0.5);
		// printf("%d %d %lf %lf\n", global->m_x, global->m_y, dir_x, dir_y);


		double x_a = global->x_bound.a;
		double x_b = global->x_bound.b;
		double y_a = global->y_bound.a;
		double y_b = global->y_bound.b;

		double dir_x = (1.0 * global->m_x / global->w_size);
		double dir_y = (1.0 * global->m_y / global->w_size);
		double p_x = x_a + (x_b - x_a) * dir_x;
		double p_y = y_a + (y_b - y_a) * dir_y;
	
		printf("%d %d %lf %lf\n", global->m_x, global->m_y, dir_x, dir_y);

		float zoom = 1.2;
		if (global->m_wheel > 0) {
			zoom = 0.9;
		} else {
			zoom = 1.2;
		}

		global->x_bound.a = (x_b + x_a - zoom * (x_b - x_a)) / 2.0;
		global->x_bound.b = (x_b + x_a + zoom * (x_b - x_a)) / 2.0;
		global->y_bound.a = (y_b + y_a - zoom * (y_b - y_a)) / 2.0;
		global->y_bound.b = (y_b + y_a + zoom * (y_b - y_a)) / 2.0;

		x_a = global->x_bound.a;
		x_b = global->x_bound.b;
		y_a = global->y_bound.a;
		y_b = global->y_bound.b;


		global->x_bound.a = p_x - (x_b - x_a) * (dir_x);
		global->x_bound.b = p_x + (x_b - x_a) * (1.0 - dir_x);
		global->y_bound.a = p_y - (y_b - y_a) * (dir_y);
		global->y_bound.b = p_y + (y_b - y_a) * (1.0 - dir_y);


		// global->x_bound.a += dir_x;
		// global->x_bound.b += dir_x;
		// global->y_bound.a += dir_y;
		// global->y_bound.b += dir_y;

		// global->x_bound.a = -2 / global->zoom;
		// global->x_bound.b = 2 / global->zoom;
		// global->y_bound.a = -2 / global->zoom;
		// global->y_bound.b = 2 / global->zoom;
		global->step_x = (global->x_bound.b - global->x_bound.a) / global->w_size;
		global->step_y = (global->y_bound.b - global->y_bound.a) / global->w_size;
	}
}

void	fractol_loop(t_global *global)
{
	SDL_Event e;
	struct timeval t1;
	struct timeval t2;

	while (!global->quit)
	{
		gettimeofday(&t1, NULL);
		fractol_frame(global);
		gettimeofday(&t2, NULL);
		printf("---> time %ld\n", (t2.tv_sec * 1000000 + t2.tv_usec) - (t1.tv_sec * 1000000 + t1.tv_usec));
		global->m_wheel = 0;
		while (SDL_PollEvent(&e) != 0)
		{
			if (e.type == SDL_QUIT)
				global->quit = 1;
			else if (e.type == SDL_MOUSEWHEEL)
				on_mouse_wheel(e.wheel.y, global);
			else if (e.type == SDL_MOUSEMOTION)
				on_mouse_move(e.motion.x, e.motion.y, global);
		}
		perform_events(global);
	}
}


t_global	*init_prog(void)
{
	t_global	*global;

	global = (t_global *)malloc(sizeof(t_global));
	if (!global)
		return (NULL);
	global->quit = 0;

	global->zoom = 1;
	global->x_bound.a = -2;
	global->x_bound.b = 2;
	global->y_bound.a = -2;
	global->y_bound.b = 2;


	set_colors(global->color_map);
	
	hipMalloc(&(global->d_color_map), 16 * 4);
	
	hipMemcpy(global->d_color_map, ((global->color_map)), 16 * 4, hipMemcpyHostToDevice);
	global->f = NULL;
	return (global);
}

void    *init_sdl(t_global *global)
{
	if (SDL_Init(SDL_INIT_VIDEO) < 0)
		return NULL;
  SDL_Window* window = SDL_CreateWindow(
    TITLE, SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED,
    global->w_size, global->w_size, SDL_WINDOW_SHOWN);
  if (window == NULL)
    return NULL;
  global->win = window;
  global->win_surface = SDL_GetWindowSurface(global->win);
	global->gpu_surface.w = global->win_surface->w;
	global->gpu_surface.h = global->win_surface->h;
	global->gpu_surface.pitch = global->win_surface->pitch;
	global->gpu_surface.BytesPerPixel = global->win_surface->format->BytesPerPixel;
	global->gpu_surface.pixels = NULL;
	global->gpu_surface.pixel_size = global->gpu_surface.pitch * global->gpu_surface.h;

	hipMalloc(&(global->gpu_surface.pixels), global->w_size * global->w_size * 4);

  return global;
}

int	main(int ac, char **av)
{
	t_global		*global;

	if (ac < 3)
	{
		printf("check args\n");
		exit(1);
	}
	global = init_prog();
	if (!global)
		return (1);
	global->w_size = atoi(av[1]);
	global->depth = atoi(av[2]);
	global->step_x = (global->x_bound.b - global->x_bound.a) / global->w_size;
	global->step_y = (global->y_bound.b - global->y_bound.a) / global->w_size;
	if (!init_sdl(global))
		return (1);
	fractol_loop(global);
	return (0);
}